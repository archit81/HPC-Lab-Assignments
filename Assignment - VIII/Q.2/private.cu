using namespace std;

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>

__global__ void
my_kernel(double *a, double *c, int n, int k)
{
    int id = threadIdx.x;
    double private_arr[1024];
    for (int i = 0; i < n; i++)
    {
        private_arr[i] = a[i];
    }

    c[id] = private_arr[id];
    for (int j = 1; j <= k; j++)
    {
        if (id + j < n)
            c[id] += private_arr[id + j];
        if (id - j >= 0)
            c[id] += private_arr[id - j];
    }
}

int main()
{
    // Size of vectors
    cout << "\n*******Neighbour addition using CUDA private memory:*******\n";
    int n = 10, k = 3;
    size_t size = n * sizeof(double);
    double *ha, *hc;
    double *da, *dc;

    ha = (double *)malloc(size);
    hc = (double *)malloc(size);

    hipMalloc(&da, size);
    hipMalloc(&dc, size);

    cout << "\nInitial Vector is:\n";
    for (int i = 0; i < n; i++)
    {
        ha[i] = rand() % 100;
        cout << ha[i] << " ";
    }

    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);

    my_kernel<<<1, 10, n>>>(da, dc, n, k);

    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

    cout << "\nResult matrix after Neighbour addition is:\n";
    for (int i = 0; i < n; i++)
    {
        cout << hc[i] << " ";
    }
    cout << "\n";

    hipFree(da);
    hipFree(dc);

    free(ha);
    free(hc);

    return 0;
}
