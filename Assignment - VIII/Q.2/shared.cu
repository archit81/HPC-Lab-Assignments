using namespace std;

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>

__global__ void
my_kernel(double *a, double *c, int n, int k)
{
    int id = threadIdx.x;
    extern __shared__ double shared_arr[];
    shared_arr[id] = a[id];
    __syncthreads();

    c[id] = shared_arr[id];
    for (int j = 1; j <= k; j++)
    {
        if (id + j < n)
            c[id] += shared_arr[id + j];
        if (id - j >= 0)
            c[id] += shared_arr[id - j];
    }
}

int main()
{
    // Size of vectors
    cout << "\n*******Neighbour addition using CUDA shared memory:*******\n";
    int n = 10, k = 3;
    size_t size = n * sizeof(double);
    double *ha, *hc;
    double *da, *dc;

    ha = (double *)malloc(size);
    hc = (double *)malloc(size);

    hipMalloc(&da, size);
    hipMalloc(&dc, size);

    cout << "\nInitial Vector is:\n";
    for (int i = 0; i < n; i++)
    {
        ha[i] = rand() % 100;
        cout << ha[i] << " ";
    }

    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);

    my_kernel<<<1, 10, n>>>(da, dc, n, k);

    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

    cout << "\nResult matrix after Neighbour addition is:\n";
    for (int i = 0; i < n; i++)
    {
        cout << hc[i] << " ";
    }
    cout << "\n";

    hipFree(da);
    hipFree(dc);

    free(ha);
    free(hc);

    return 0;
}
