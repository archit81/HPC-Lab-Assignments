using namespace std;

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>

__global__ void
my_kernel(double *a, double *c, int n, int k)
{
    int id = threadIdx.x;
    c[id] = a[id];
    for (int j = 1; j <= k; j++)
    {
        if (id + j < n)
            c[id] += a[id + j];
        if (id - j >= 0)
            c[id] += a[id - j];
    }
}

int main()
{
    // Size of vectors
    cout << "\n*******Neighbour addition using CUDA global memory:*******\n";
    int n = 10, k = 3;
    size_t size = n * sizeof(double);
    double *ha, *hc;
    double *da, *dc;

    ha = (double *)malloc(size);
    hc = (double *)malloc(size);

    hipMalloc(&da, size);
    hipMalloc(&dc, size);

    cout << "\nInitial Vector is:\n";
    for (int i = 0; i < n; i++)
    {
        ha[i] = rand() % 100;
        cout << ha[i] << " ";
    }

    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);

    my_kernel<<<1, 10>>>(da, dc, n, k);

    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

    cout << "\nResult matrix after Neighbour addition is:\n";
    for (int i = 0; i < n; i++)
    {
        cout << hc[i] << " ";
    }
    cout << "\n";

    hipFree(da);
    hipFree(dc);

    free(ha);
    free(hc);

    return 0;
}
