using namespace std;

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>

__global__ void
matrixMultiplication(double *a, double *b, double *c, int n)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row < n && col < n)
        for (int i = 0; i < n; i++)
            c[row * n + col] += a[row * n + i] * b[i * n + col];
}

int main()
{
    // Size of vectors
    cout << "Enter dimension of matrix:\n";
    int n;
    cin >> n;
    size_t size = n * n * sizeof(double);
    double *ha, *hb, *hc;
    double *da, *db, *dc;

    ha = (double *)malloc(size);
    hb = (double *)malloc(size);
    hc = (double *)malloc(size);

    hipMalloc(&da, size);
    hipMalloc(&db, size);
    hipMalloc(&dc, size);

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            ha[i * n + j] = rand() % 100;
            hb[i * n + j] = rand() % 100;
        }
    }

    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16), gridDim(ceil((float)n / blockDim.x), ceil((float)n / blockDim.y));
    matrixMultiplication<<<gridDim, blockDim>>>(da, db, dc, n);

    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

    cout << "\nMatrix A is:\n";
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << ha[i * n + j] << " ";
        }
        cout << "\n";
    }
    cout << "\nMatrix B is:\n";
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << hb[i * n + j] << " ";
        }
        cout << "\n";
    }
    cout << "\nResult matrix after multiplication is:\n";
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << hc[i * n + j] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    free(ha);
    free(hb);
    free(hc);

    return 0;
}
