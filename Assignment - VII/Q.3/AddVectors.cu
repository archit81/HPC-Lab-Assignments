using namespace std;

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>

// CUDA addition kernel
__global__ void
vectorAdd(double *a, double *b, double *c, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] + b[id];
}

int main(int argc, char *argv[])
{
    // Size of vectors
    cout << "Enter size of vectors:\n";
    int n;
    cin >> n;
    size_t size = n * sizeof(double);
    // Host input & output vectors
    double *ha, *hb, *hc;
    // Device input & output vectors
    double *da, *db, *dc;

    ha = (double *)malloc(size);
    hb = (double *)malloc(size);
    hc = (double *)malloc(size);
    cout << "Enter elements of vector-a:\n";
    for (int i = 0; i < n; i++)
        cin >> ha[i];

    cout << "Enter elements of vector-b:\n";
    for (int i = 0; i < n; i++)
        cin >> hb[i];

    hipMalloc(&da, size);
    hipMalloc(&db, size);
    hipMalloc(&dc, size);

    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256, numBlocks = (int)ceil((float)n / threadsPerBlock);
    vectorAdd<<<numBlocks, threadsPerBlock>>>(da, db, dc, n);

    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

    cout << "\nResult Vector after addition is:\n";
    for (int i = 0; i < n; i++)
    {
        cout << hc[i] << " ";
    }
    cout << "\n";

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    free(ha);
    free(hb);
    free(hc);

    return 0;
}
