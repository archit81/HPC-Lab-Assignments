using namespace std;

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>

__global__ void
fibonacci(int n, int *res)
{
    // nth fibonacci formula = [[(√5 + 1)/2] ^ n] / √5
    int threadID = threadIdx.x;
    double phi = (1 + sqrt(5.0)) / 2;
    res[threadID] = round(pow(phi, threadID) / sqrt(5.0));
}

int main()
{
    // Size of vectors
    cout << "Enter length of fibonacci series:\n";
    int n;
    cin >> n;

    size_t size = n * sizeof(int);
    int *h_arr;
    int *d_arr;

    h_arr = (int *)malloc(size);

    hipMalloc(&d_arr, size);

    fibonacci<<<1, n>>>(n, d_arr);

    hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);

    cout << "\nFibonacci Series is:\n";
    for (int i = 0; i < n; i++)
    {
        cout << h_arr[i] << " ";
    }

    hipFree(d_arr);

    free(h_arr);

    return 0;
}
